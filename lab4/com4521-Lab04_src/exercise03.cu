#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define N 2048
#define M N
#define THREADS_PER_BLOCK 256

void checkCUDAError(const char*);
void random_ints(int *a);

__global__ void matrixAdd(int *a, int *b, int *c, int max) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < max )
	{
	    c[i] = a[i] + b[i];
	}
}

void matrixAddCPU(int *a, int *b, int *c)
{
    
	for(int i=0;i<N*M;i++)
	{
	    c[i] = a[i] + b[i];
	}
}

int validate(int* c_left,int* c_right)
{
    int total_error_count = 0;
	for(int i=0;i<N*M;i++)
	{   
	    if(c_left[i] != c_right[i])
		{
		    printf("error! left : %d , right : %d \n",c_left[i],c_right[i]);
		    total_error_count = total_error_count + 1;
		}
	}
	return total_error_count;
}


int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * M * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc \n");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	
	c_ref = (int *)malloc(size);

    matrixAddCPU(a,b,c_ref);
	
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy \n");



	// Launch add() kernel on GPU
	matrixAdd << <(N * M) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(d_a, d_b, d_c, N*M);
	hipDeviceSynchronize();
	checkCUDAError("CUDA kernel \n");


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	errors = validate(c,c_ref);
	printf("Total error's reported : %d \n",errors);
	checkCUDAError("CUDA memcpy v2 \n");

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup \n");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for(int i=0;i<N*M;i++)
	{
	    a[i] = rand();
	}
}
