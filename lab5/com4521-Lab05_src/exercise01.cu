#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define N 65536
#define THREADS_PER_BLOCK 128

__device__ int data_a[N] , data_b[N] , data_c[N];


void checkCUDAError(const char*);
void random_ints(int *a);



__global__ void vectorAdd(int *a, int *b, int *c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

#define GIGABYTES_CONV (1024*1024)

int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * sizeof(int);
    hipEvent_t start , stop;
	hipDeviceProp_t device_prop;
    int active_device = 0 , num_devs = 0;
	hipGetDeviceCount(&num_devs);
	printf(" number cuda devices : %d \n" , num_devs);
	
	hipGetDevice(&active_device);
	hipGetDeviceProperties(&device_prop,active_device);
	
	long mem_bus_width = device_prop.memoryBusWidth; 
	int mem_clock_rate = device_prop.memoryClockRate;
	float mem_clock_rate_gbps = mem_clock_rate / GIGABYTES_CONV; 
	float memory_bandwidth = mem_clock_rate_gbps * mem_bus_width;
	
	printf(" mem_bus_width : %ld , mem_clock_rate : %d , theoretical_memory_bandwidth : %f \n" , mem_bus_width,mem_clock_rate,memory_bandwidth);
	
hipEventCreate(&start);
hipEventCreate(&stop);

// Get symbol addresses of static copies in CUDA
    hipGetSymbolAddress((void **)&d_a, data_a);
    hipGetSymbolAddress((void **)&d_b, data_b);
    hipGetSymbolAddress((void **)&d_c, data_c);


	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	hipMemcpyToSymbol(HIP_SYMBOL(data_a),a,size);
	hipMemcpyToSymbol(HIP_SYMBOL(data_b),b,size);
	
	// Copy inputs to device
	checkCUDAError("CUDA memcpy symbol (to) ");

hipEventRecord(start);
	// Launch add() kernel on GPU
	vectorAdd << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(d_a, d_b, d_c, N);
hipEventRecord(stop);
hipEventSynchronize(stop);

	checkCUDAError("CUDA kernel");

float milliseconds = 0;
hipEventElapsedTime(&milliseconds,start,stop);

    printf("cuda time delta for kernel : %f \n",milliseconds);
    float computed_bw = ((N * 3 * sizeof(int) * 8 * 1000 ) / (milliseconds * 1024 * 1024 * 1024)); 
	printf(" real_computed_bandwidth = %f \n" , computed_bw);

	// Copy result back to host
	hipMemcpyFromSymbol(c, HIP_SYMBOL(data_c), size);
	checkCUDAError("CUDA memcpy symbol (from) ");

	// Cleanup
	free(a); free(b); free(c);

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = rand();
	}
}
