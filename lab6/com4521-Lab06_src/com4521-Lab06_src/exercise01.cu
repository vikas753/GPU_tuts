#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""


#define A_WIDTH 1024
#define A_HEIGHT 1024
#define B_WIDTH 1024
#define B_HEIGHT 1024
#define C_WIDTH B_WIDTH
#define C_HEIGHT A_HEIGHT

#define BLOCK_SIZE 8
#define NUM_SUBS (A_WIDTH / BLOCK_SIZE)

__device__ float d_A[A_HEIGHT][A_WIDTH];
__device__ float d_B[B_HEIGHT][B_WIDTH];
__device__ float d_C[C_HEIGHT][C_WIDTH];

float h_A[A_HEIGHT][A_WIDTH];
float h_B[B_HEIGHT][B_WIDTH];
float h_C[C_HEIGHT][C_WIDTH];
float h_C_ref[C_HEIGHT][C_WIDTH];

__constant__ int block_size_const = 0;

void checkCUDAError(const char *msg);
void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[B_HEIGHT][B_WIDTH], float C[C_HEIGHT][C_WIDTH]);
int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH]);

__global__ void matrixMulCUDA()
{
    // Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int x = bx*BLOCK_SIZE + tx;
	int y = by*BLOCK_SIZE + ty;
    

	float Csub = 0;
	//iterate A_WIDTH (same as B_HEIGHT) to calculate the product
	for (int k = 0; k < A_WIDTH; k++){
		Csub += d_A[y][k] * d_B[k][x]; 
	}

	// Store the product value of C matrix
	d_C[y][x] = Csub;
}

int requiredSM(int block_size)
{
    return (2 * sizeof(float) * block_size * block_size);
}


__global__ void matrixMulCUDASharedMemory()
{
    //Define some shared memory for a sub block of matrices A an B
    extern __shared__ float shared_mem_data[];
    float* As = (float*)&shared_mem_data[0];
	float* Bs = (float*)&shared_mem_data[block_size_const * block_size_const];
	

	// Block index
	int bx = blockIdx.x;
	int b_dim_x = blockDim.x;
	int by = blockIdx.y;
	int b_dim_y = blockDim.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
    
	//Running sum of product of A and B matrices
    float Csub = 0;
 
	//iterate through the number of sub matrices of A and B
	for (int i = 0; i < NUM_SUBS; i++){
		//TODO: Calculate indices of A and B matrix required to load the shared block of memory
        int a_x = tx + i*b_dim_x;
		int a_y = ty + b_dim_y*by;
		int b_x = tx + b_dim_x*bx;
		int b_y = i*b_dim_y + ty;
        
        
        if((a_x < A_WIDTH) & (a_y < A_HEIGHT))
        {
            As[ty*block_size_const + tx] = d_A[a_y][a_x];
        }
		else
		{
		    As[ty*block_size_const + tx] = 0;		
		}
		
		if((b_x < B_WIDTH) & (b_y < B_HEIGHT))
		{
		    Bs[ty*block_size_const + tx] = d_B[b_y][b_x];
		}
		else
		{
		    Bs[ty*block_size_const + tx] = 0;
		}
			
        			
		
        // Sync to ensure sub matrix is fully loaded
		__syncthreads();
        
        //TODO: sum products of A and B sub matrices
		for (int k = 0; k < block_size_const; ++k)
		{
		    Csub = Csub + As[ty*block_size_const + k]*Bs[k*block_size_const + tx];
		}
        
        // Sync to prevent run ahead (blocks loading new SM values before others have completed)
		__syncthreads();
        
	}

    //TODO: caluclate the indices of sub matrix C
	int c_x = tx + b_dim_x*bx;
	int c_y = ty + b_dim_y*by;
    
	if((c_x < C_WIDTH) & (c_y < C_HEIGHT))
	{
	// Store the product value of C matrix
	    d_C[c_y][c_x] = Csub;
	}
	else
	{
	    d_C[c_y][c_x] = 0;
	}
}


int main(int argc, char **argv)

{
	unsigned int mem_size_A, mem_size_B, mem_size_C;
	unsigned int x, y, errors;
	int maxActiveBlocks;
	float msec, occupancy;
	hipDeviceProp_t props;
	hipEvent_t start, stop;

    int nDevices;
    hipGetDeviceCount(&nDevices);
	
	hipGetDeviceProperties(&props, 0);
	printf(" Cuda device count : %d \n" , nDevices);

	if (A_WIDTH != B_HEIGHT){
		printf("Error: A_HEIGHT and B_WIDTH do not match\n");
	}

	mem_size_A = sizeof(float)* A_WIDTH* A_HEIGHT;
	mem_size_B = sizeof(float)* B_WIDTH* B_HEIGHT;
	mem_size_C = sizeof(float)* C_WIDTH* C_HEIGHT;

	// Initialise A
	for (y = 0; y < A_HEIGHT; y++)
	for (x = 0; x <A_WIDTH; x++)
		h_A[y][x] = (float)rand() / RAND_MAX;
	// Initialise B
	for (y = 0; y < B_HEIGHT; y++)
	for (x = 0; x <B_WIDTH; x++)
		h_B[y][x] = (float)rand() / RAND_MAX;


	// copy host memory to device
	hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, mem_size_A);
	hipMemcpyToSymbol(HIP_SYMBOL(d_B), h_B, mem_size_B);
	checkCUDAError("CUDA memcpy");

	// Allocate CUDA events that we'll use for timing
	hipEventCreate(&start);
	hipEventCreate(&stop);
	checkCUDAError("CUDA event creation");

	// Setup execution parameters
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(C_WIDTH / BLOCK_SIZE, C_HEIGHT / BLOCK_SIZE);
	hipEventRecord(start);
	
    
    matrixMulCUDA << < grid, threads >> >();
	int minGridSize = 0;
    int block_size = BLOCK_SIZE;
    hipOccupancyMaxPotentialBlockSize(&minGridSize,&block_size,matrixMulCUDASharedMemory,0);
    printf(" block_size : %d with Variable SMEM \n",block_size);
	hipMemcpyToSymbol(HIP_SYMBOL(block_size_const), &block_size, sizeof(int));
	int required_SM = requiredSM(block_size);
    //TODO: Comment out the above line and complete the shared memory version of the kernel
    matrixMulCUDASharedMemory << < grid, threads, required_SM >> >();
    
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	checkCUDAError("CUDA kernel execution and timing");

	hipEventElapsedTime(&msec, start, stop);
	hipDeviceSynchronize();
	checkCUDAError("CUDA timing");

	// Compute the ocupancy
	occupancy = (props.maxBlocksPerMultiProcessor * (block_size*block_size))/(props.maxThreadsPerMultiProcessor * props.multiProcessorCount);

	// Copy result from device to host
	hipMemcpyFromSymbol(h_C, HIP_SYMBOL(d_C), mem_size_C);
	checkCUDAError("CUDA memcpy results");

	// Compute reference CPU version
	matrixMulCPU(h_A, h_B, h_C_ref);

	// Check for errors
	errors = matrixMulTest(h_C, h_C_ref);
	if (errors)
		printf("%d total errors\n", errors);
	else
		printf("Test passed successfully\n");

    printf(" props.maxBlocksPerMultiProcessor : %d , props.maxThreadsPerMultiProcessor : %d , props.multiProcessorCount : %d \n" , props.maxBlocksPerMultiProcessor , props.maxThreadsPerMultiProcessor , props.multiProcessorCount); 

	printf("Kernel time was %f with theoretical occupancy of %f\n", msec, occupancy);

}


void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[C_HEIGHT][C_WIDTH], float C[C_HEIGHT][C_WIDTH])
{
	int x, y, k;
	for (y = 0; y < C_HEIGHT; y++){
		for (x = 0; x < C_WIDTH; x++){
			C[y][x] = 0;
			for (k = 0; k < A_WIDTH; k++){
				C[y][x] += A[y][k] * B[k][x];
			}
		}
	}

}

int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH])
{
	int errors = 0;
	int y, x;

	for (y = 0; y < C_HEIGHT; y++){
		for (x = 0; x < C_WIDTH; x++){
			if (C[y][x] != Cref[y][x]){
				errors++;
				printf("Device item c[%d][%d] = %f does not mach host result %f\n", y, x, C[y][x], Cref[y][x]);
			}
		}
	}

	return errors;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
