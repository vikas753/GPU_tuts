#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// include kernels and cuda headers after definitions of structures
#include "kernels.cuh" 


void checkCUDAError(const char*);
void readRecords(student_record *records);

void maximumMark_atomic(student_records*, student_records*, student_records*, student_records*);
void maximumMark_recursive(student_records*, student_records*, student_records*, student_records*);
void maximumMark_SM(student_records*, student_records*, student_records*, student_records*);
void maximumMark_shuffle(student_records*, student_records*, student_records*, student_records*);


int main(void) {
	student_record *recordsAOS;
	student_records *h_records;
	student_records *h_records_result;
	student_records *d_records;
	student_records *d_records_result;
	
	//host allocation
	recordsAOS = (student_record*)malloc(sizeof(student_record)*NUM_RECORDS);
	h_records = (student_records*)malloc(sizeof(student_records));
	h_records_result = (student_records*)malloc(sizeof(student_records));

	//device allocation
	hipMalloc((void**)&d_records, sizeof(student_records));
	hipMalloc((void**)&d_records_result, sizeof(student_records));
	checkCUDAError("CUDA malloc");
    
	printf( " Reading records  ! \n");
	//read file
	readRecords(recordsAOS);


	//Exercise 1.1) Convert recordsAOS to a structure of arrays in h_records
	for(int i=0;i<NUM_RECORDS;i++)
	{
	    h_records->student_ids[i] = recordsAOS[i].student_id;
	    h_records->assignment_marks[i] = recordsAOS[i].assignment_mark;
	}
	
	//free AOS as it is no longer needed
	free(recordsAOS);
    printf( " Firing Kernels ! \n");

	//apply each approach in turn 
	maximumMark_atomic(h_records, h_records_result, d_records, d_records_result);
	maximumMark_recursive(h_records, h_records_result, d_records, d_records_result);
	maximumMark_SM(h_records, h_records_result, d_records, d_records_result);
	maximumMark_shuffle(h_records, h_records_result, d_records, d_records_result);


	// Cleanup
	free(h_records);
	free(h_records_result);
	hipFree(d_records);
	hipFree(d_records_result);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void readRecords(student_record *records){
	FILE *f = NULL;
	f = fopen("com4521_large.dat", "rb"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find com4521_large.dat file \n");
		exit(1);
	}

	//read student data
	if (fread(records, sizeof(student_record), NUM_RECORDS, f) != NUM_RECORDS){
		fprintf(stderr, "Error: Unexpected end of file!\n");
		exit(1);
	}
	fclose(f);
}


void maximumMark_atomic(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("1) CUDA memcpy");

	hipEventRecord(start, 0);

	//find highest mark using GPU
	dim3 blocksPerGrid(NUM_RECORDS / THREADS_PER_BLOCK, 1, 1);
	dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
	maximumMark_atomic_kernel << <blocksPerGrid, threadsPerBlock >> >(d_records);

	hipDeviceSynchronize();
	checkCUDAError("Atomics: CUDA kernel");

	// Copy result back to host
	hipMemcpyFromSymbol(&max_mark, HIP_SYMBOL(d_max_mark), sizeof(float));
	hipMemcpyFromSymbol(&max_mark_student_id, HIP_SYMBOL(d_max_mark_student_id), sizeof(int));
	checkCUDAError("Atomics: CUDA memcpy back");

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("Atomics: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//Exercise 2)
void maximumMark_recursive(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	int i , threadsPerBlockVal = THREADS_PER_BLOCK;
	float max_mark = 0.0f;
	int max_mark_student_id = 0;
	student_records *d_records_temp;
	float time;
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Recursive: CUDA memcpy");

	hipEventRecord(start, 0);
	
	//Exercise 2.3) Recursively call GPU steps until there are THREADS_PER_BLOCK values left
	
	//find highest mark using GPU
	dim3 blocksPerGrid(NUM_RECORDS / threadsPerBlockVal, 1, 1);
	int num_records = NUM_RECORDS;
	
	for(;threadsPerBlockVal > 1;threadsPerBlockVal = threadsPerBlockVal >> 1)
	{
 	    dim3 threadsPerBlock(threadsPerBlockVal, 1, 1);
	    maximumMark_recursive_kernel << <blocksPerGrid, threadsPerBlock >> >(d_records,d_records_result);
	    hipDeviceSynchronize();
	    checkCUDAError(" CUDA recursive kernel");
        num_records = num_records >> 1;
	    d_records = d_records_result;
	}
	


	//Exercise 2.4) copy back the final THREADS_PER_BLOCK values
    hipMemcpy(h_records_result, d_records_result, sizeof(student_records), hipMemcpyDeviceToHost);

	//Exercise 2.5) reduce the final THREADS_PER_BLOCK values on CPU
    
	for(int i=0;i<NUM_RECORDS;i=i+1)
	{
	    if(h_records_result->assignment_marks[i] > max_mark)
		{
		    max_mark = h_records_result->assignment_marks[i];
			max_mark_student_id = h_records_result->student_ids[i];
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output the result
	printf("Recursive: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//Exercise 3)
void maximumMark_SM(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i, threadsPerBlockVal = THREADS_PER_BLOCK;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("SM: CUDA memcpy");

	hipEventRecord(start, 0);
	//find highest mark using GPU
	dim3 blocksPerGrid(NUM_RECORDS / threadsPerBlockVal, 1, 1);
	int num_records = NUM_RECORDS;
	
	//Exercise 3.4) Call the shared memory reduction kernel
	for(;threadsPerBlockVal > 1;threadsPerBlockVal = threadsPerBlockVal >> 1)
	{
 	    dim3 threadsPerBlock(threadsPerBlockVal, 1, 1);
	    maximumMark_SM_kernel << <blocksPerGrid, threadsPerBlock >> >(d_records,d_records_result);
	    hipDeviceSynchronize();
	    checkCUDAError(" CUDA recursive kernel");
        num_records = num_records >> 1;
	    d_records = d_records_result;
	}
	
	//Exercise 3.5) Copy the final block values back to CPU
    hipMemcpy(h_records_result, d_records_result, sizeof(student_records), hipMemcpyDeviceToHost);


	//Exercise 3.6) Reduce the block level results on CPU
	for(int i=0;i<NUM_RECORDS;i=i+1)
	{
	    if(h_records_result->assignment_marks[i] > max_mark)
		{
		    max_mark = h_records_result->assignment_marks[i];
			max_mark_student_id = h_records_result->student_ids[i];
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("SM: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//Exercise 4)
void maximumMark_shuffle(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i;
	unsigned int warps_per_grid;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	unsigned threadsPerBlockVal = THREADS_PER_BLOCK;
	
	
	max_mark = 0;
	max_mark_student_id = 0.0f;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Shuffle: CUDA memcpy");
	
	hipEventRecord(start, 0);

	//Exercise 4.2) Execute the kernel, copy back result, reduce final values on CPU

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("SM: CUDA memcpy");

	hipEventRecord(start, 0);
	//find highest mark using GPU
	dim3 blocksPerGrid(NUM_RECORDS / threadsPerBlockVal, 1, 1);
	int num_records = NUM_RECORDS;
	
	//Exercise 4.3) Call the shared memory reduction kernel
	for(;threadsPerBlockVal > 1;threadsPerBlockVal = threadsPerBlockVal >> 1)
	{
 	    dim3 threadsPerBlock(threadsPerBlockVal, 1, 1);
	    maximumMark_shuffle_kernel << <blocksPerGrid, threadsPerBlock >> >(d_records,d_records_result);
	    hipDeviceSynchronize();
	    checkCUDAError(" CUDA recursive kernel");
        num_records = num_records >> 1;
	    d_records = d_records_result;
	}
	
	//Exercise 4.4) Copy the final block values back to CPU
    hipMemcpy(h_records_result, d_records_result, sizeof(student_records), hipMemcpyDeviceToHost);


	//Exercise 4.5) Reduce the block level results on CPU
	for(int i=0;i<NUM_RECORDS;i=i+1)
	{
	    if(h_records_result->assignment_marks[i] > max_mark)
		{
		    max_mark = h_records_result->assignment_marks[i];
			max_mark_student_id = h_records_result->student_ids[i];
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("Shuffle: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}